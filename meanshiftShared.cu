/*Antoniadis Moschos, AEM = 8761, AUTH 2018, sharedMemory version*/
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <omp.h>
#include <sys/time.h>
#include <float.h>
#include <hip/hip_runtime.h>

#define EPSILON 0.0000001
#define MAX_ITERATIONS 30
#define N 2
#define SIGMA 1
#define BLOCK_SIZE 128
#define INPUT_FILE "x.bin"
#define RESULTS_FILE "results.txt"
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

/********* Device Function - Gaussian ******/
__device__ double gaussian(double norm)
{

    return exp(-norm / (2 * SIGMA * SIGMA));
}

/******** Kernel Function *******/
__global__ void meanshiftKernel(double *devX_, double *devY, int size)
{
    extern __shared__ double sharedBuffer[];
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < size) //if idx bigger than matrix height return
    {
        double *devX = &sharedBuffer[0];
        int j, k, iterations = 0;
        for (k = 0; k < N; k++)
            devX[idx * N + k] = devX_[idx * N + k];
        __syncthreads();

        double sumNum[N] = {0}, sumDenum = 0, dist, meanshift = 0xFFFFFFFF, m_new[N], m[N];

        for (k = 0; k < N; k++)
            m[k] = devY[idx * N + k];

        while (meanshift > EPSILON && iterations < MAX_ITERATIONS)
        {
            meanshift = 0;

            for (k = 0; k < N; k++)
                sumNum[k] = 0;
            sumDenum = 0;

            for (j = 0; j < size; j++)
            {
                dist = 0;
                for (k = 0; k < N; k++)
                    dist += pow(m[k] - devX[j * N + k], 2);
                if (dist < SIGMA) //dist is already squared
                {
                    for (k = 0; k < N; k++)
                        sumNum[k] += gaussian(dist) * devX[j * N + k];
                    sumDenum += gaussian(dist);
                }
            }
            for (k = 0; k < N; k++)
            {
                m_new[k] = sumNum[k] / sumDenum;

                meanshift += pow(m_new[k] - m[k], 2);

                m[k] = m_new[k];
            }
            meanshift = sqrt(meanshift);
            iterations++;

            if (idx == 0)
                printf("Iteration %d  error = %.9f \n", iterations, meanshift); //print iterations for first point
        }
        for (k = 0; k < N; k++)
        {
            devY[idx * N + k] = m[k];
        }
        __syncthreads();
    }
}

/******** Main ********/

int main(int argc, char **argv)
{
    FILE *inFile, *resultsFile;
    int filesize;
    size_t totalsize;
    double *x, *y;         // host matrices
    double *dev_x, *dev_y; // GPU matrices
    hipEvent_t start, stop;
    float elapsedTime;
    /*Input file open */
    if ((inFile = fopen(INPUT_FILE, "r+")) == NULL)
    {
        printf("\nFile not found\n");
        exit(1);
    }

    /* Get the size of the file */
    fseek(inFile, 0L, SEEK_END);
    filesize = ftell(inFile) / (sizeof(double) * N); //find the number of points
    rewind(inFile);
    totalsize = filesize * N * sizeof(double); //size in bytes
    printf("\nTotal points = %d  Dimensions = %d\n", filesize, N);

    /*Malloc for input buffer*/
    if ((x = (double *)malloc(totalsize)) == NULL)
        exit(1);

    /*Malloc for output buffer*/
    if ((y = (double *)malloc(totalsize)) == NULL)
        exit(1);

    /*Read the data to buffer*/
    if ((fread(x, sizeof(double), filesize * N, inFile)) != filesize * N)
    {
        fprintf(stderr, "Unable to read data\n");
        exit(1);
    }
    fclose(inFile);

    /*Malloc for GPU input buffer*/
    gpuErrchk(hipMalloc((void **)&dev_x, totalsize));

    /*Malloc for output buffer*/
    gpuErrchk(hipMalloc((void **)&dev_y, totalsize));

    /*Copy data to GPU global memory*/
    gpuErrchk(hipMemcpy(dev_x, x, totalsize, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(dev_y, x, totalsize, hipMemcpyHostToDevice));

    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (filesize + threadsPerBlock - 1) / threadsPerBlock;

    hipEventCreate(&start);
    hipEventRecord(start, 0);

    meanshiftKernel<<<blocksPerGrid, threadsPerBlock, filesize * N * sizeof(double)>>>(dev_x, dev_y, filesize);

    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Elapsed time : %f ms\n", elapsedTime);

    gpuErrchk(hipMemcpy(y, dev_y, totalsize, hipMemcpyDeviceToHost));

    hipFree(dev_x);
    hipFree(dev_y);

    /* Write results to file */
    resultsFile = fopen(RESULTS_FILE, "wb");
    for (int i = 0; i < filesize; i++)
    {
        for (int k = 0; k < N; k++)
            fprintf(resultsFile, "%.6f\t", y[i * N + k]);
        fprintf(resultsFile, "\n");
    }
    fclose(resultsFile);

    free(x);
    free(y);
}